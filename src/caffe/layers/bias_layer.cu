#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

// CUDA kernel for forward
template <typename Dtype>
__global__ void BiasForward(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype* bias_data, const int div_factor) {

  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out[index] = in[index] + bias_data[c];
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void BiasBackward(const int n, const Dtype* in_diff, Dtype* out_diff) {

  CUDA_KERNEL_LOOP(index, n) {
    
    out_diff[index] = in_diff[index];
  }
}

// CUDA kernel for element-wise parameter backward
template <typename Dtype>
__global__ void BiasParamBackward(const int n, const Dtype* in_diff, Dtype* out_diff) {

  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index];
  }
}

template <typename Dtype>
void BiasLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();
  const Dtype* bias_data = this->blobs_[0]->gpu_data();
  const int div_factor = channel_shared_ ? channels : 1;

  // NOLINT_NEXT_LINE(whitespace/operators)
  BiasForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, channels, dim, bottom_data, top_data, bias_data, div_factor);
  CUDA_POST_KERNEL_CHECK;

  //LOG(INFO) << "gpu:: bias layer";
}

template <typename Dtype>
void BiasLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {

  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();

  // Propagate to param
  if (this->param_propagate_down_[0]) {

  	Dtype* bias_diff = this->blobs_[0]->mutable_gpu_diff();
  	// bias_diff is set as 0, then accumulated over batches
  	//caffe_gpu_set<Dtype>(this->blobs_[0]->count(), Dtype(0), bias_diff);
  	
    int cdim = channels * dim;
  	Dtype dsum = 0.;

  	for (int n = 0; n < bottom[0]->num(); ++n) {

  		Dtype* temp_buff = multiplier_.mutable_gpu_diff();

  		// compute element-wise diff
  		// NOLINT_NEXT_LINE(whitespace/operators)
  		BiasParamBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
  			cdim, top_diff + top[0]->offset(n), multiplier_.mutable_gpu_diff());
  		CUDA_POST_KERNEL_CHECK;

  		// I do not have a single clue about what the hell happens here
  		if (channel_shared_) {
  			Dtype d;
  			caffe_gpu_dot<Dtype>(channels * dim, multiplier_.gpu_diff(), multiplier_.gpu_data(), &d);
  			dsum += d;

  		} else {
  			caffe_gpu_gemv<Dtype>(CblasNoTrans, channels, dim, 1.,
            multiplier_.gpu_diff(), multiplier_.gpu_data(), 1., bias_diff);

  		}

  	}	//end for loop

  	if (channel_shared_) {
  		//caffe_gpu_set(this->blobs_[0]->count(), Dtype(dsum), bias_diff);
      caffe_gpu_add_scalar(this->blobs_[0]->count(), Dtype(dsum), bias_diff);
  	}

  }

  // Propagate to bottom
  if (propagate_down[0]) {

  	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  	// NOLINT_NEXT_LINE(whitespace/operators)
  	BiasBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
  		count, top_diff, bottom_diff);

  	CUDA_POST_KERNEL_CHECK;
  }

}

INSTANTIATE_LAYER_GPU_FUNCS(BiasLayer);

} 	// namespace caffe
